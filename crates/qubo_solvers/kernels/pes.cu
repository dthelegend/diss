#include "hip/hip_runtime.h"
#include "qubo_solvers.h"

__device__
void flip_each(
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* deltas,
    qubo_t* solution_list,
    qubo_t* eval,
    const size_t i
) {
    *eval += deltas[i];
    for (size_t j = 0; j < i; j++) {
        deltas[j] += 2 * qubo_problem[j * problem_size + i] * (2 * solution_list[i] - 1) * (2 * solution_list[j] - 1);
    }
    deltas[i] = - deltas[i];
    solution_list[i] = 1 - solution_list[i];
}

__global__
void flip(
    const size_t n,
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* deltas,
    qubo_t* solution_list,
    qubo_t* eval_list,
    const size_t alpha
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        flip_each(
            problem_size,
            qubo_problem,
            deltas + (i * problem_size),
            solution_list + (i * problem_size),
            eval_list + i,
            alpha
        );
    }
}

void search(
    const size_t n,
    const int num_blocks,
    const int block_size,
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* best_solution,
    qubo_t* best_evaluation,
    qubo_t* deltas,
    qubo_t* solution_list,
    qubo_t* eval_list,
    const size_t i
) {
    if (i == 0) {
        // Check all values for minimum
        // This could in theory be parallelised to log n, but the constant factor tends to be relatively small
        size_t min_eval_index = 0;

        for (size_t j = 1; j < n; j++) {
            if (eval_list[j] < eval_list[min_eval_index]) {
                min_eval_index = j;
            }
        }

        hipDeviceSynchronize();

        if (eval_list[min_eval_index] < *best_evaluation) {
            hipMemcpyAsync(best_evaluation, eval_list + min_eval_index, sizeof(qubo_t), hipMemcpyDeviceToHost);
            hipMemcpyAsync(best_solution, solution_list + (min_eval_index * problem_size), problem_size * sizeof(qubo_t), hipMemcpyDeviceToHost);
        }

        return;
    }

    // search left
    search(
        n,
        num_blocks,
        block_size,
        problem_size,
        qubo_problem,
        best_solution,
        best_evaluation,
        deltas,
        solution_list,
        eval_list,
        i - 1
    );

    flip<<<num_blocks, block_size>>>(
        n,
        problem_size,
        qubo_problem,
        deltas,
        solution_list,
        eval_list,
        i - 1
    );

    search(
        n,
        num_blocks,
        block_size,
        problem_size,
        qubo_problem,
        best_solution,
        best_evaluation,
        deltas,
        solution_list,
        eval_list,
        i - 1
    );

    flip<<<num_blocks, block_size>>>(
        n,
        problem_size,
        qubo_problem,
        deltas,
        solution_list,
        eval_list,
        i - 1
    );
}

extern "C" void run_pes_solver(
    const int block_size,
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* best_solution,
    qubo_t* best_evaluation,
    const qubo_t* deltas,
    const qubo_t* solution_list,
    const qubo_t* eval_list,
    const size_t i
) {
    int n = 1 << (problem_size - i - 1);
    int num_blocks = (n + block_size - 1) / block_size;

    // TODO Throw an exception if the memory cannot b alloc'd

    qubo_t* cuda_qubo_problem;
    qubo_t* cuda_deltas;
    qubo_t* cuda_solution_list;
    qubo_t* cuda_eval_list;

    hipMallocManaged(&cuda_qubo_problem, problem_size * problem_size * sizeof(qubo_t));
    hipMallocManaged(&cuda_deltas, n * problem_size * sizeof(qubo_t));
    hipMallocManaged(&cuda_solution_list, n * problem_size * sizeof(qubo_t));
    hipMallocManaged(&cuda_eval_list, n * sizeof(qubo_t));

    hipMemcpyAsync(cuda_qubo_problem, qubo_problem, problem_size * problem_size * sizeof(qubo_t), hipMemcpyHostToDevice);
    hipMemcpyAsync(cuda_deltas, deltas, n * problem_size * sizeof(qubo_t), hipMemcpyHostToDevice);
    hipMemcpyAsync(cuda_solution_list, solution_list, n * problem_size * sizeof(qubo_t), hipMemcpyHostToDevice);
    hipMemcpyAsync(cuda_eval_list, eval_list, n * sizeof(qubo_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    search(
        n,
        num_blocks,
        block_size,
        problem_size,
        cuda_qubo_problem,
        best_solution,
        best_evaluation,
        cuda_deltas,
        cuda_solution_list,
        cuda_eval_list,
        i
    );
}
