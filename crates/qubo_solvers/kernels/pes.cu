#include "hip/hip_runtime.h"
#include "qubo_solvers.h"
#include <iostream>

const size_t MAX_DEPTH = 16;
const size_t MAX_PROBLEM_SIZE = MAX_DEPTH;
const size_t MAX_SUB_PROBLEMS = 1 << (MAX_DEPTH - 1);

__device__ size_t n_sub_problems;

__device__ size_t d_problem_size;
__device__ qubo_t d_qubo_problem[MAX_PROBLEM_SIZE][MAX_PROBLEM_SIZE];

__device__ qubo_t d_curr_solution[MAX_PROBLEM_SIZE];
__device__ qubo_t d_curr_deltas[MAX_SUB_PROBLEMS][MAX_PROBLEM_SIZE];
__device__ qubo_t d_curr_eval[MAX_SUB_PROBLEMS];

__device__ size_t d_min_solution_index = 0;
__device__ qubo_t d_min_solution[MAX_PROBLEM_SIZE];
__device__ qubo_t d_min_eval;

__global__ void __launch_bounds__(MAX_PROBLEM_SIZE, MAX_SUB_PROBLEMS) flip_part_one(
    size_t i
) {
    size_t sub_problem_start_index = blockIdx.x * blockDim.x;
    int sub_problem_stride = blockDim.x * gridDim.x;

    size_t j = threadIdx.x;

    for (
        auto sub_problem_index = sub_problem_start_index;
        sub_problem_index < n_sub_problems;
        sub_problem_index += sub_problem_stride
    ) {
        d_curr_deltas[sub_problem_index][i] += 2 * d_qubo_problem[i][j] * (2 * d_curr_solution[i] - 1) * (2 * d_curr_solution[j] - 1);
    }
}

__global__ void __launch_bounds__(MAX_PROBLEM_SIZE, MAX_SUB_PROBLEMS) flip_part_two(
    size_t i
) {
    size_t sub_problem_start_index = blockIdx.x * blockDim.x + threadIdx.x;
    int sub_problem_stride = blockDim.x * gridDim.x;

    if (sub_problem_start_index == 0) {
        d_curr_solution[i] = 1 - d_curr_solution[i];
    }

    for (
        auto sub_problem_index = sub_problem_start_index;
        (sub_problem_index < n_sub_problems) && (threadIdx.x == 0);
        sub_problem_index += sub_problem_stride
    ) {
        d_curr_eval[sub_problem_index] += d_curr_deltas[sub_problem_index][i];
        d_curr_deltas[sub_problem_index][i] = - d_curr_deltas[sub_problem_index][i];
    }
}

__global__ void __launch_bounds__(MAX_PROBLEM_SIZE, MAX_PROBLEM_SIZE) set_best_solution(
    size_t sub_problem_index
) {
//     size_t start_index = blockIdx.x * blockDim.x + threadIdx.x;
//     int stride = blockDim.x * gridDim.x;
//
//     if (start_index == 0) {
//         d_min_solution_index = sub_problem_index;
//     }
//
//     for (size_t index = start_index; index < d_problem_size; index += stride) {
//         d_curr_solution[index] = d_curr_solution[index];
//     }
}

__global__ void
__launch_bounds__(1, 1)
flip(
    size_t i
) {
    if (i > 0) {
        flip_part_one<<<i,n_sub_problems>>>(i);
    }
    // TODO maybe increase thread use by calculating this
    flip_part_two<<<1,n_sub_problems>>>(i);
}

__global__ void
__launch_bounds__(1, 1)
search(
    size_t i
) {
    printf("%d\n", i);
    if (i == 0) {
//         size_t min_eval_index = 0;
//
//         for (size_t j = 1; j < n_sub_problems; j++) {
//             printf("%d", d_curr_eval[j]);
//             if (d_curr_eval[j] < d_curr_eval[min_eval_index]) {
//                 min_eval_index = j;
//             }
//             printf("\n");
//         }
//
//         if (d_curr_eval[min_eval_index] < d_min_eval) {
//             set_best_solution<<<d_problem_size, 1>>>(min_eval_index);
//         }
    } else {
//         search<<<1,1>>>(i - 1);

        flip<<<1,1>>>(i - 1);

//         search<<<1,1>>>(i - 1);

        flip<<<1,1>>>(i - 1);
    }
}

extern "C" hipError_t run_pes_solver(
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* best_solution,
    qubo_t* best_evaluation,
    const qubo_t* solution_list,
    const qubo_t* deltas_list,
    const qubo_t* eval_list,
    const size_t i
) {
    size_t n = 1 << (problem_size - i - 1);

    std::cout << "Hello from CUDA! 0" << std::endl;
    cudaCheckError(hipMemcpy(&n_sub_problems, &n, sizeof(size_t), hipMemcpyDefault));
    std::cout << "Hello from CUDA! 1" << std::endl;

    cudaCheckError(hipMemcpy(&d_problem_size, &problem_size, sizeof(size_t), hipMemcpyDefault));
    cudaCheckError(hipMemcpy2D(d_qubo_problem, MAX_PROBLEM_SIZE * sizeof(qubo_t), qubo_problem, problem_size * sizeof(qubo_t), i * sizeof(qubo_t), i, hipMemcpyDefault));

    std::cout << "Hello from CUDA! 2" << std::endl;
    cudaCheckError(hipMemcpy2D(d_curr_solution, MAX_PROBLEM_SIZE * sizeof(qubo_t), solution_list, problem_size * sizeof(qubo_t), i * sizeof(qubo_t), n, hipMemcpyDefault));
    cudaCheckError(hipMemcpy2D(d_curr_deltas, MAX_PROBLEM_SIZE * sizeof(qubo_t), deltas_list, problem_size * sizeof(qubo_t), i * sizeof(qubo_t), n, hipMemcpyDefault));
    cudaCheckError(hipMemcpy(d_curr_eval, eval_list, sizeof(qubo_t) * n, hipMemcpyDefault));

    std::cout << "Hello from CUDA! 3" << std::endl;
    cudaCheckError(hipMemcpy(d_min_solution, best_solution, sizeof(qubo_t) * problem_size, hipMemcpyDefault));
    cudaCheckError(hipMemcpy(&d_min_eval, best_evaluation, sizeof(qubo_t), hipMemcpyDefault));

    std::cout << "Hello from CUDA! 4" << std::endl;

    search<<<1,1>>>(i);

    std::cout << "Hello from CUDA! 5" << std::endl;

    cudaCheckError(hipGetLastError());

    std::cout << "Hello from CUDA! 6" << std::endl;

    hipDeviceSynchronize();

    std::cout << "Goodbye from CUDA! 7" << std::endl;

    cudaCheckError(hipMemcpy(best_solution, d_min_solution, sizeof(qubo_t) * problem_size, hipMemcpyDefault));
    cudaCheckError(hipMemcpy(best_evaluation, &d_min_eval, sizeof(qubo_t), hipMemcpyDefault));

    return hipSuccess;
}
