
#include <hip/hip_runtime.h>
#include <iostream>

typedef int32_t qubo_t;

__device__
void flip_each(
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* deltas,
    qubo_t* solution_list,
    qubo_t* eval,
    const size_t i
) {
    *eval += deltas[i];
    for (size_t j = 0; j < i; j++) {
        deltas[i] += qubo_problem[j * problem_size + i] * (2 * solution_list[i] - 1) * (2 * solution_list[j] - 1);
    }
    deltas[i] = -deltas[i];
    solution_list[i] = 1 - solution_list[i];
}

__global__
void flip(
    const size_t n,
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* deltas,
    qubo_t* solution_list,
    qubo_t* eval_list,
    const size_t alpha
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        flip_each(
            problem_size,
            qubo_problem,
            deltas + (i * problem_size),
            solution_list + (i * problem_size),
            eval_list + i,
            alpha
        );
    }
}

__host__ void search(
    const size_t n,
    const int num_blocks,
    const int block_size,
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* best_solution,
    qubo_t* best_evaluation,
    qubo_t* deltas,
    qubo_t* solution_list,
    qubo_t* eval_list,
    const size_t i
) {
    if (i == 0) {
        // Check all values for minimum
        // This could in theory be parallelised to log n, but the constant factor tends to be relatively small
        for (size_t i = 0; i < n; i++) {
            if (eval_list[i] < *best_evaluation) {
                *best_evaluation = eval_list[i];
                std::cout << "Wowie from " << i << std::endl;
                for (int j = 0; j < problem_size; j++)
                    std::cout << (solution_list + (i * problem_size))[j];
                std::cout << std::endl;

                hipMemcpy(best_solution, solution_list + (i * problem_size), problem_size, hipMemcpyDefault);
            }
        }
        return;
    }
    
    // search left
    search(
        n,
        num_blocks,
        block_size,
        problem_size,
        qubo_problem,
        best_solution,
        best_evaluation,
        deltas,
        solution_list,
        eval_list,
        i - 1
    );

    flip<<<num_blocks, block_size>>>(
        n,
        problem_size,
        qubo_problem,
        deltas,
        solution_list,
        eval_list,
        i - 1
    );

    hipDeviceSynchronize();

    search(
        n,
        num_blocks,
        block_size,
        problem_size,
        qubo_problem,
        best_solution,
        best_evaluation,
        deltas,
        solution_list,
        eval_list,
        i - 1
    );

    flip<<<num_blocks, block_size>>>(
        n,
        problem_size,
        qubo_problem,
        deltas,
        solution_list,
        eval_list,
        i - 1
    );

    hipDeviceSynchronize();
}

extern "C" void run_pes_solver(
    const int block_size,
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* best_solution,
    qubo_t* best_evaluation,
    const qubo_t* deltas,
    const qubo_t* solution_list,
    const qubo_t* eval_list,
    const size_t i
) {
    std::cout << "Hello from CUDA!" << std::endl;

    int n = 1 << (problem_size - i - 1);
    int num_blocks = (n + block_size - 1) / block_size;

    qubo_t* cuda_qubo_problem;
    hipMallocManaged(&cuda_qubo_problem, problem_size * problem_size * sizeof(qubo_t));
    hipMemcpy(cuda_qubo_problem, qubo_problem, problem_size * problem_size * sizeof(qubo_t), hipMemcpyHostToDevice);

    qubo_t* cuda_deltas;
    hipMallocManaged(&cuda_deltas, n * problem_size * sizeof(qubo_t));
    hipMemcpy(cuda_deltas, deltas, n * problem_size * sizeof(qubo_t), hipMemcpyHostToDevice);

    qubo_t* cuda_solution_list;
    hipMallocManaged(&cuda_solution_list, n * problem_size * sizeof(qubo_t));
    hipMemcpy(cuda_solution_list, solution_list, n * problem_size * sizeof(qubo_t), hipMemcpyHostToDevice);

    qubo_t* cuda_eval_list;
    hipMallocManaged(&cuda_eval_list, n * sizeof(qubo_t));
    hipMemcpy(cuda_eval_list, eval_list, n * sizeof(qubo_t), hipMemcpyHostToDevice);

    search(
        n,
        num_blocks,
        block_size,
        problem_size,
        cuda_qubo_problem,
        best_solution,
        best_evaluation,
        cuda_deltas,
        cuda_solution_list,
        cuda_eval_list,
        i
    );

    std::cout << "Goodbye from CUDA!" << std::endl;
}