
#include <hip/hip_runtime.h>
#include <iostream>

typedef int32_t qubo_t;

__device__
void flip_each(
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* deltas,
    qubo_t* solution_list,
    qubo_t* eval,
    const size_t i
) {
    *eval += deltas[i];
    for (size_t j = 0; j < i; j++) {
        deltas[j] += 2 * qubo_problem[j * problem_size + i] * (2 * solution_list[i] - 1) * (2 * solution_list[j] - 1);
    }
    deltas[i] = - deltas[i];
    solution_list[i] = 1 - solution_list[i];
}

__global__
void flip(
    const size_t n,
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* deltas,
    qubo_t* solution_list,
    qubo_t* eval_list,
    const size_t alpha
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        flip_each(
            problem_size,
            qubo_problem,
            deltas + (i * problem_size),
            solution_list + (i * problem_size),
            eval_list + i,
            alpha
        );
    }
}

__host__ void search(
    const size_t n,
    const int num_blocks,
    const int block_size,
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* best_solution,
    qubo_t* best_evaluation,
    qubo_t* deltas,
    qubo_t* solution_list,
    qubo_t* eval_list,
    const size_t i
) {
    if (i == 0) {
        // Check all values for minimum
        // This could in theory be parallelised to log n, but the constant factor tends to be relatively small
        size_t min_eval_index = 0;

        for (size_t j = 1; j < n; j++) {
            if (eval_list[j] < eval_list[min_eval_index]) {
                min_eval_index = j;
            }
        }

        if (eval_list[min_eval_index] < *best_evaluation) {
            hipMemcpy(best_evaluation, eval_list + min_eval_index, sizeof(qubo_t), hipMemcpyDeviceToHost);
            hipMemcpy(best_solution, solution_list + (min_eval_index * problem_size), problem_size * sizeof(qubo_t), hipMemcpyDeviceToHost);
        }

        return;
    }
    
    // search left
    search(
        n,
        num_blocks,
        block_size,
        problem_size,
        qubo_problem,
        best_solution,
        best_evaluation,
        deltas,
        solution_list,
        eval_list,
        i - 1
    );

    flip<<<num_blocks, block_size>>>(
        n,
        problem_size,
        qubo_problem,
        deltas,
        solution_list,
        eval_list,
        i - 1
    );

    hipDeviceSynchronize();

    search(
        n,
        num_blocks,
        block_size,
        problem_size,
        qubo_problem,
        best_solution,
        best_evaluation,
        deltas,
        solution_list,
        eval_list,
        i - 1
    );

    flip<<<num_blocks, block_size>>>(
        n,
        problem_size,
        qubo_problem,
        deltas,
        solution_list,
        eval_list,
        i - 1
    );

    hipDeviceSynchronize();
}

extern "C" void run_pes_solver(
    const int block_size,
    const size_t problem_size,
    const qubo_t* qubo_problem,
    qubo_t* best_solution,
    qubo_t* best_evaluation,
    const qubo_t* deltas,
    const qubo_t* solution_list,
    const qubo_t* eval_list,
    const size_t i
) {
    std::cout << "Hello from CUDA!" << std::endl;

    int n = 1 << (problem_size - i - 1);
    int num_blocks = (n + block_size - 1) / block_size;
    
    // TODO Throw an exception if the memory cannot b alloc'd
    
    qubo_t* cuda_qubo_problem;
    hipMallocManaged(&cuda_qubo_problem, problem_size * problem_size * sizeof(qubo_t));
    hipMemcpy(cuda_qubo_problem, qubo_problem, problem_size * problem_size * sizeof(qubo_t), hipMemcpyHostToDevice);

    qubo_t* cuda_deltas;
    hipMallocManaged(&cuda_deltas, n * problem_size * sizeof(qubo_t));
    hipMemcpy(cuda_deltas, deltas, n * problem_size * sizeof(qubo_t), hipMemcpyHostToDevice);

    qubo_t* cuda_solution_list;
    hipMallocManaged(&cuda_solution_list, n * problem_size * sizeof(qubo_t));
    hipMemcpy(cuda_solution_list, solution_list, n * problem_size * sizeof(qubo_t), hipMemcpyHostToDevice);

    qubo_t* cuda_eval_list;
    hipMallocManaged(&cuda_eval_list, n * sizeof(qubo_t));
    hipMemcpy(cuda_eval_list, eval_list, n * sizeof(qubo_t), hipMemcpyHostToDevice);

    search(
        n,
        num_blocks,
        block_size,
        problem_size,
        cuda_qubo_problem,
        best_solution,
        best_evaluation,
        cuda_deltas,
        cuda_solution_list,
        cuda_eval_list,
        i
    );

    std::cout << "Goodbye from CUDA!" << std::endl;
}